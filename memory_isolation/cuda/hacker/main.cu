
#include <hip/hip_runtime.h>
#include <stdio.h>

#define ARRAY_LEN 4096
#define RUN_COUNT 1000

int max_print=20;
unsigned long long fnd_count=0;

void checker(int round, char* buf)
{
	int i;

	for(i=0; i<ARRAY_LEN; i++)
	{   
		switch(buf[i])
		{
			case 'A':
			case 'B':
			case 'C':
			case 'D':
			case 'E':
			case 'F':
			case 'G':
			case 'H':
			case 'I':
			case 'J':
				if(max_print)
				{
					printf("%d\t%d\t%c\n", round, i, buf[i]);
					max_print--;
				}
				fnd_count++;
				break;

			default:
				break;
		}
	}   
}

int main()
{
	int i;
	size_t buf_size = ARRAY_LEN*sizeof(char);
	char *h_buf, *d_buf;

	hipSetDevice(0);

	for(i=0; i<RUN_COUNT; i++)
	{
		h_buf = (char*)malloc(buf_size);
		hipMalloc( (void**)&d_buf, buf_size ); 

		hipMemcpy( h_buf, d_buf, buf_size, hipMemcpyDeviceToHost ); 

		checker(i, h_buf);

		free(h_buf);
		hipFree( d_buf );
	}

	printf("Total find %llu data\n", fnd_count);

	return 0;
}
