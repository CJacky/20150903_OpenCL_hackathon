
#include <hip/hip_runtime.h>
#include <stdio.h>

#define STR_NUM 1024 
#define RUN_COUNT 1

__global__ void func(char *buf) 
{
	int idx = gridDim.x*blockIdx.x;
	buf[idx  ] = 'A';
	buf[idx+1] = 'B';
	buf[idx+2] = 'C';
	buf[idx+3] = 'D';
	buf[idx+4] = 'E';
	buf[idx+5] = 'F';
	buf[idx+6] = 'G';
	buf[idx+7] = 'H';
	buf[idx+8] = 'I';
	buf[idx+9] = 'J';
}

int main()
{
	int i;
	size_t buf_size = STR_NUM*10*sizeof(char);
	char *h_buf, *d_buf;

	hipSetDevice(0);

	for(i=0; i<RUN_COUNT; i++)
	{
		h_buf = (char*)malloc(buf_size);
		hipMalloc( (void**)&d_buf, buf_size ); 

		func<<<STR_NUM, 10>>>(d_buf);
		hipMemcpy( h_buf, d_buf, buf_size, hipMemcpyDeviceToHost ); 

		free(h_buf);
		hipFree( d_buf );
	}
	return 0;
}
